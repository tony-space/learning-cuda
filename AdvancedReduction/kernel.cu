#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <numeric>
#include <algorithm>
#include <cstdlib>
#include <climits>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <chrono>

constexpr size_t kNumbers = 704185371;
constexpr size_t kMaxBlockSize = 512;

constexpr int kDefaultValue = 0;

template<typename T>
static inline __device__ __host__ T divCeil(T a, T b)
{
	return (a - 1) / b + 1;
}

static inline __device__ int warpReduce(int val)
{
	for (int offset = warpSize / 2; offset > 0; offset >>= 1)
	{
		auto neighbour = __shfl_down_sync(0xFFFFFFFF, val, offset);
		val += neighbour;
	}
	return val;
}

__global__ void reduceKernel(int* values, size_t size, int* output)
{
	int val = kDefaultValue;

	auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
	auto warpId = threadIdx.x / warpSize;
	auto laneId = threadIdx.x % warpSize;
	auto gridSize = blockDim.x * gridDim.x;

	extern __shared__ int cache[];
	int cacheSize = divCeil<int>(blockDim.x, warpSize); //equals to amount of warps in blocks

	if (threadId < size)
		val = values[threadId];
	if (threadId + gridSize < size)
		val = val + values[threadId + gridSize];

	val = warpReduce(val);
	if (laneId == 0)
		cache[warpId] = val;

	if (warpId > 0)
		return;

	__syncthreads();

	val = laneId < cacheSize ? cache[laneId] : kDefaultValue;
	val = warpReduce(val);

	if (laneId == 0)
		output[blockIdx.x] = val;
}

int main(int argc, char** argv)
{
	srand(42);
	thrust::host_vector<int> hostNumbers;
	hostNumbers.reserve(kNumbers);
	for (size_t i = 0; i < kNumbers; ++i)
		hostNumbers.push_back(rand() % 101 - 50);

	auto begin = std::chrono::high_resolution_clock::now();
	int controlResult = std::accumulate(hostNumbers.begin(), hostNumbers.end(), 0);
	auto end = std::chrono::high_resolution_clock::now();
	printf("Control result: %d\r\n", controlResult);
	printf("Elapsed time on CPU: %.3f ms\r\n", float(std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count()) * 1e-6f);

	thrust::device_vector<int> deviceNumbers = hostNumbers;
	thrust::device_vector<int> intermediate(divCeil(divCeil(kNumbers, size_t(2)), kMaxBlockSize));
	auto x = intermediate.size();

	hipError_t status;

	hipEvent_t start;
	status = hipEventCreate(&start);
	assert(status == hipSuccess);

	hipEvent_t stop;
	status = hipEventCreate(&stop);
	assert(status == hipSuccess);

	status = hipEventRecord(start);
	assert(status == hipSuccess);


	auto buffer1 = thrust::raw_pointer_cast(deviceNumbers.data());
	auto buffer2 = thrust::raw_pointer_cast(intermediate.data());

	for (size_t numbers = kNumbers; numbers > 1;)
	{
		size_t pairs = divCeil(numbers, size_t(2));
		size_t warps = divCeil(pairs, size_t(32));
		dim3 blockSize(min(kMaxBlockSize, warps * size_t(32)));
		dim3 gridSize(divCeil(pairs, size_t(blockSize.x)));

		reduceKernel <<<gridSize, blockSize, blockSize.x / 32>>> (buffer1, numbers, buffer2);
		std::swap(buffer1, buffer2);
		numbers = gridSize.x;
	}

	status = hipEventRecord(stop);
	assert(status == hipSuccess);

	int result;
	status = hipMemcpy(&result, buffer1, sizeof(result), hipMemcpyDeviceToHost);
	assert(status == hipSuccess);

	status = hipEventSynchronize(stop);
	assert(status == hipSuccess);

	float ms;
	status = hipEventElapsedTime(&ms, start, stop);
	assert(status == hipSuccess);

	printf("Elapsed time on GPU: %.3f ms\r\n", ms);
	printf("Result: %d \r\n", result);

	status = hipEventDestroy(stop);
	assert(status == hipSuccess);
	status = hipEventDestroy(start);
	assert(status == hipSuccess);

	assert(controlResult == result);

	system("pause");
	return 0;
}