#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <numeric>
#include <algorithm>
#include <cstdlib>
#include <climits>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <chrono>

constexpr size_t kNumbers = 704185371;
//constexpr size_t kNumbers = 704181;
constexpr size_t kMaxBlockSize = 256;

constexpr int kDefaultValue = 0;

template<typename T>
static inline __device__ __host__ T divCeil(T a, T b)
{
	return (a - 1) / b + 1;
}

template<typename T>
static inline __device__ T warpReduce(T val)
{
	for (auto offset = warpSize >> 1; offset > 0; offset >>= 1)
	{
		T neighbour = __shfl_down_sync(0xFFFFFFFF, val, offset);
		val += neighbour;
	}
	return val;
}

__global__ void reduceKernel(const int* __restrict__ values, size_t size, int* __restrict__ output)
{
	auto val = kDefaultValue;

	auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
	auto warpId = threadIdx.x / unsigned(warpSize);
	auto laneId = threadIdx.x % warpSize;
	auto gridSize = blockDim.x * gridDim.x;

	extern __shared__ int cache[];
	auto cacheSize = divCeil(blockDim.x, unsigned(warpSize)); //equals to amount of warps in blocks

	if (threadId < size)
		val = values[threadId];
	if (threadId + gridSize < size)
		val = val + values[threadId + gridSize];

	val = warpReduce(val);
	if (laneId == 0)
		cache[warpId] = val;

	if (warpId > 0)
		return;

	__syncthreads();

	val = laneId < cacheSize ? cache[laneId] : kDefaultValue;
	val = warpReduce(val);

	if (laneId == 0)
		output[blockIdx.x] = val;
}

int main(int argc, char** argv)
{
	srand(42);
	thrust::host_vector<int> hostNumbers;
	hostNumbers.reserve(kNumbers);
	for (size_t i = 0; i < kNumbers; ++i)
		hostNumbers.push_back(rand() % 101 - 50);

	auto begin = std::chrono::high_resolution_clock::now();
	auto controlResult = std::accumulate(hostNumbers.begin(), hostNumbers.end(), 0);
	auto end = std::chrono::high_resolution_clock::now();
	printf("Control result: %d\r\n", controlResult);
	printf("Elapsed time on CPU: %.3f ms\r\n", float(std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count()) * 1e-6f);

	thrust::device_vector<int> deviceNumbers = hostNumbers;
	thrust::device_vector<int> intermediate(divCeil(divCeil(kNumbers, size_t(2)), kMaxBlockSize));
	auto x = intermediate.size();

	hipError_t status;

	hipEvent_t start;
	status = hipEventCreate(&start);
	assert(status == hipSuccess);

	hipEvent_t stop;
	status = hipEventCreate(&stop);
	assert(status == hipSuccess);

	status = hipEventRecord(start);
	assert(status == hipSuccess);


	auto buffer1 = thrust::raw_pointer_cast(deviceNumbers.data());
	auto buffer2 = thrust::raw_pointer_cast(intermediate.data());

	for (size_t numbers = kNumbers; numbers > 1;)
	{
		size_t pairs = divCeil(numbers, size_t(2));
		size_t warps = divCeil(pairs, size_t(32));
		dim3 blockSize(min(kMaxBlockSize, warps * size_t(32)));
		dim3 gridSize(divCeil(pairs, size_t(blockSize.x)));

		reduceKernel <<<gridSize, blockSize, blockSize.x / 32 * sizeof(int)>>> (buffer1, numbers, buffer2);
		//hipDeviceSynchronize();
		std::swap(buffer1, buffer2);
		numbers = gridSize.x;
	}

	status = hipEventRecord(stop);
	assert(status == hipSuccess);

	int result;
	status = hipMemcpy(&result, buffer1, sizeof(result), hipMemcpyDeviceToHost);
	assert(status == hipSuccess);

	status = hipEventSynchronize(stop);
	assert(status == hipSuccess);

	float ms;
	status = hipEventElapsedTime(&ms, start, stop);
	assert(status == hipSuccess);

	printf("Elapsed time on GPU: %.3f ms\r\n", ms);
	printf("Result: %d\r\n", result);
	deviceNumbers = hostNumbers;
	printf("Result2: %d\r\n", thrust::reduce(deviceNumbers.begin(), deviceNumbers.end()));

	status = hipEventDestroy(stop);
	assert(status == hipSuccess);
	status = hipEventDestroy(start);
	assert(status == hipSuccess);

	assert(controlResult == result);

	system("pause");
	return 0;
}