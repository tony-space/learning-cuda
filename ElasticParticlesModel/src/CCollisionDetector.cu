#include "hip/hip_runtime.h"
#include <>

#include "CCollisionDetector.hpp"
#include "CSimulation.hpp"
//#include <hip/hip_runtime.h>

template<typename T>
static inline __device__ __host__ T sqr(T x)
{
	return x * x;
}


__global__ void copyResultKernel(
	const hipcub::KeyValuePair<int, float>* particlesReductionResult,
	const hipcub::KeyValuePair<int, float>* wallsReductionResult,
	size_t particles,
	size_t walls,
	SObjectsCollision* __restrict__ collisionResult)
{
	auto particlesResult = *particlesReductionResult;
	auto wallsResult = *wallsReductionResult;

	SObjectsCollision result;

	if (particlesResult.value < wallsResult.value)
	{
		const size_t matIndex = size_t(particlesResult.key);
		auto i = matIndex / particles;
		auto j = matIndex % particles;

		result.collisionType = SObjectsCollision::CollisionType::ParticleToParticle;
		result.object1 = i;
		result.object2 = j;
		result.predictedTime = particlesResult.value;
	}
	else
	{
		const size_t matIndex = size_t(wallsResult.key);
		auto wall = matIndex / particles;
		auto particle = matIndex % particles;

		result.collisionType = SObjectsCollision::CollisionType::ParticleToPlane;
		result.object1 = particle;
		result.object2 = wall;
		result.predictedTime = wallsResult.value;
	}

	*collisionResult = result;
}

__global__ void predictParticleParticleCollisionsKernel(const SParticleSOA particles, float* __restrict__ matrix)
{
	auto i = blockIdx.y * blockDim.y + threadIdx.y;
	auto j = blockIdx.x * blockDim.x + threadIdx.x;
	auto threadId = i * particles.count + j;

	if (i >= particles.count || j >= particles.count)
		return;

	float result = INFINITY;

	if (i > j)
	{
		auto selfPos = particles.pos[i];
		auto selfVel = particles.vel[i];
		auto selfRad = particles.radius[i];

		auto otherPos = particles.pos[j];
		auto otherVel = particles.vel[j];
		auto otherRad = particles.radius[j];

		//Let's solve a quadratic equation to predict the exact collision time.
		//The quadric equation can be get from the following vector equation:
		//(R1 + V1 * dt) - (R2 + V2 * dt) = rad1 + rad2  : the distance between new positions equals the sum of two radii
		//where R1 and R2 are radius vectors of the current particles position
		//      V1 and V2 are velocity vectors
		//      rad1 and rad2 are particles' radii
		//      dt is the unknown variable
		//Vector dot product satisfies a distributive law.

		float3 deltaR = selfPos - otherPos;
		float3 deltaV = selfVel - otherVel;

		//Quadratic equation coefficients
		float a = dot(deltaV, deltaV);
		float b = 2.0f * dot(deltaR, deltaV);
		float c = dot(deltaR, deltaR) - sqr(selfRad + otherRad);
		float discriminant = sqr(b) - 4.0f * a * c;

		//if particles don't move relatively each other (deltaV = 0)
		if (fabsf(a) <= 0.0f)
		{
			matrix[threadId] = INFINITY;
			return;
		}
		//if particles are flying away
		if (b > 0.0f)
		{
			matrix[threadId] = INFINITY;
			return;
		}
		//if particles somehow have already penetrated one each other (e.g. due to incorrect position generation or numerical errors)
		if (c < 0.0f)
		{
			matrix[threadId] = 0.0f;
			return;
		}
		//if particles ways never intersect
		if (discriminant < 0.0f)
		{
			matrix[threadId] = INFINITY;
			return;
		}

		float sqrtD = sqrtf(discriminant);
		//Here is a tricky part.
		//You might think, why we even need to compute dt2 if it definitely is greater than dt1?
		//The answer is these two values can be negative, which means two contacts has already been somewhere in the past.
		float dt1 = (-b - sqrtD) / (2.0f * a);
		float dt2 = (-b + sqrtD) / (2.0f * a);

		if (dt2 >= 0.0f)
			result = dt2;
		if (dt1 >= 0.0f)
			result = dt1;
	}
	matrix[threadId] = result;
}

__global__ void predictParticlePlaneCollisionsKernel(
	const SParticleSOA particles,
	const SPlane* __restrict__ planes,
	const size_t planesCount,
	float* __restrict__ matrix)
{
	auto planeId = blockIdx.y * blockDim.y + threadIdx.y;
	auto particleId = blockIdx.x * blockDim.x + threadIdx.x;
	auto threadId = planeId * particles.count + particleId;

	if (planeId >= planesCount || particleId >= particles.count)
		return;

	auto pos = particles.pos[particleId];
	auto vel = particles.vel[particleId];
	auto rad = particles.radius[particleId];
	auto plane = planes[planeId];
	float result = INFINITY;

	auto velProjection = dot(plane.normal, vel);
	if (velProjection < 0.0f)
		result = plane.Distance(pos, rad) / -velProjection;

	matrix[threadId] = result;
}

CCollisionDetector::ArgMinReduction::ArgMinReduction(size_t rows, size_t columns)
{
	m_reductionResult = thrust::device_malloc<hipcub::KeyValuePair<int, float>>(1);

	m_matrix.resize(rows * columns);

	size_t tempStorageBytesSize = 0;
	auto status = hipcub::DeviceReduce::ArgMin(nullptr, tempStorageBytesSize, m_matrix.data().get(), m_reductionResult.get(), int(m_matrix.size()));
	assert(status == hipSuccess);
	m_cubTemporaryStorage.resize(tempStorageBytesSize);
}

void CCollisionDetector::ArgMinReduction::Reduce()
{
	//thrust::host_vector<float> debugVec = m_matrix;
	//for (auto f : debugVec)
	//	printf("%.2f ", f);
	//printf("\n");

	size_t tempStorageBytesSize = m_cubTemporaryStorage.size();
	auto status = hipcub::DeviceReduce::ArgMin(m_cubTemporaryStorage.data().get(), tempStorageBytesSize, m_matrix.data().get(), m_reductionResult.get(), int(m_matrix.size()));
	assert(status == hipSuccess);

	//hipcub::KeyValuePair<int, float> debug;
	//status = hipMemcpy(&debug, m_reductionResult.get(), sizeof(debug), hipMemcpyDeviceToHost);
	//assert(status == hipSuccess);
	//printf("%d %f\n", debug.key, debug.value);
	//printf("\n");
}

CCollisionDetector::CCollisionDetector(const SParticleSOA d_particles, const thrust::host_vector<SPlane>& worldBoundaries) :
	m_deviceParticles(d_particles),
	m_devicePlanes(worldBoundaries),
	m_particle2particleReduction(d_particles.count, d_particles.count),
	m_particle2planeReduction(worldBoundaries.size(), d_particles.count)
{
	m_collisionResult = thrust::device_malloc<SObjectsCollision>(1);
}

SObjectsCollision* CCollisionDetector::FindEarliestCollision()
{
	auto particles = unsigned(m_deviceParticles.count);
	auto walls = unsigned(m_devicePlanes.size());

	dim3 blockDim;
	dim3 gridDim;

	blockDim = dim3(32u, 32u);
	gridDim = dim3(divCeil(particles, blockDim.x), divCeil(particles, blockDim.y));
	predictParticleParticleCollisionsKernel << <gridDim, blockDim >> > (m_deviceParticles, m_particle2particleReduction.m_matrix.data().get());

	blockDim = dim3(1024u, 1u);
	gridDim = dim3(divCeil(particles, blockDim.x), divCeil(walls, blockDim.y));
	predictParticlePlaneCollisionsKernel << <gridDim, blockDim >> > (m_deviceParticles, m_devicePlanes.data().get(), walls, m_particle2planeReduction.m_matrix.data().get());

	m_particle2particleReduction.Reduce();
	m_particle2planeReduction.Reduce();

	copyResultKernel << <1, 1 >> > (m_particle2particleReduction.m_reductionResult.get(), m_particle2planeReduction.m_reductionResult.get(), particles, walls, m_collisionResult.get());

	//SObjectsCollision debug;
	//auto status = hipMemcpy(&debug, m_collisionResult.get(), sizeof(debug), hipMemcpyDeviceToHost);
	//assert(status == hipSuccess);
	//printf("%f\n", debug.predictedTime);
	//printf("\n");

	return m_collisionResult.get();
}