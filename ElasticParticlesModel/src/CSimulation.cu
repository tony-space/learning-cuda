#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <cassert>
#include "CSimulation.hpp"

__device__ void resolveParticle2ParticleCollision(const float3& pos1, float3& vel1, const float3& pos2, float3& vel2)
{
	auto centerOfMassVel = (vel1 + vel2) / 2.0f;
	auto v1 = vel1 - centerOfMassVel;
	auto v2 = vel2 - centerOfMassVel;

	auto planeNormal = normalize(pos1 - pos2);

	v1 = reflect(v1, planeNormal);
	v2 = reflect(v2, planeNormal);

	vel1 = v1 + centerOfMassVel;
	vel2 = v2 + centerOfMassVel;
}

__device__ __constant__ const SObjectsCollision earilestCollision;

__global__ void moveParticlesKernel(SParticleSOA particles, float dt)
{
	auto threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= particles.count)
		return;

	dt = fminf(earilestCollision.predictedTime, dt);

	auto pos = particles.pos[threadId];
	auto vel = particles.vel[threadId];

	pos += vel * dt;
	particles.pos[threadId] = pos;
}

__global__ void resolveCollisionsKernel(
	SParticleSOA particles,
	const float dt,
	const SPlane* __restrict__ pPlanes)
{
	if (dt < earilestCollision.predictedTime)
		return;

	auto pos1 = particles.pos[earilestCollision.object1];
	auto vel1 = particles.vel[earilestCollision.object1];

	switch (earilestCollision.collisionType)
	{
	case SObjectsCollision::CollisionType::ParticleToPlane:
		vel1 = reflect(vel1, pPlanes[earilestCollision.object2].normal);
		break;

	case SObjectsCollision::CollisionType::ParticleToParticle:
		auto pos2 = particles.pos[earilestCollision.object2];
		auto vel2 = particles.vel[earilestCollision.object2];

		resolveParticle2ParticleCollision(pos1, vel1, pos2, vel2);
		particles.vel[earilestCollision.object2] = vel2;

		break;
	}

	particles.vel[earilestCollision.object1] = vel1;
}

CSimulation::CSimulation(SParticleSOA d_particles) : m_deviceParticles(d_particles)
{
	thrust::host_vector<SPlane> hostPlanes;
	hostPlanes.push_back(SPlane(make_float3(1.0, 0.0, 0.0), -0.5));
	hostPlanes.push_back(SPlane(make_float3(-1.0, 0.0, 0.0), -0.5));
	hostPlanes.push_back(SPlane(make_float3(0.0, 1.0, 0.0), -0.5));
	hostPlanes.push_back(SPlane(make_float3(0.0, -1.0, 0.0), -0.5));
	hostPlanes.push_back(SPlane(make_float3(0.0, 0.0, 1.0), -0.5));
	hostPlanes.push_back(SPlane(make_float3(0.0, 0.0, -1.0), -0.5));

	m_collisionDetector = std::make_unique<CCollisionDetector>(m_deviceParticles, hostPlanes);
}

float CSimulation::UpdateState(float dt)
{
	dim3 blockDim(64);
	dim3 gridDim((unsigned(m_deviceParticles.count) - 1) / blockDim.x + 1);

	auto d_earliestCollistion = m_collisionDetector->FindEarliestCollision();
	auto status = hipMemcpyToSymbolAsync(HIP_SYMBOL(earilestCollision), d_earliestCollistion, sizeof(SObjectsCollision), 0, hipMemcpyDeviceToDevice);
	assert(status == hipSuccess);
	moveParticlesKernel << <gridDim, blockDim >> > (m_deviceParticles, dt);
	resolveCollisionsKernel <<<1, 1 >>> (m_deviceParticles, dt, m_collisionDetector->GetPlanes());

	return dt;
}

std::unique_ptr<ISimulation> ISimulation::CreateInstance(SParticleSOA d_particles)
{
	return std::make_unique<CSimulation>(d_particles);
}