#include "hip/hip_runtime.h"
#include <GL\glew.h>

//#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cuda_gl_interop.h>
#include <cassert>
#include <hip/hip_vector_types.h>

#include "CScene.hpp"

__device__ const float3 spherePos = { 0.0f, 0.0f, -1.5f };
__device__ const float sphereRad = 1.0f;

__global__ void rayTracingKernel(hipSurfaceObject_t surfObj, unsigned width, unsigned height, float time)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	float3 ray = normalize(make_float3(float(x) / float(width - 1) * 2.0f - 1.0f, float(y) / float(height - 1) * 2.0f - 1.0f, -1.0f));

	float projection = dot(spherePos, ray);
	float3 sphereCenterProj = ray * projection;
	float3 diff = sphereCenterProj - spherePos;
	float diffLength = norm3df(diff.x, diff.y, diff.z);
	if (diffLength > sphereRad)
	{
		float4 result = { 0.0f, 0.0f, 0.0f, 1.0f };
		surf2Dwrite(result, surfObj, x * sizeof(result), y);
		return;
	}

	float halfChord = sqrtf(sphereRad * sphereRad - diffLength * diffLength);
	
	float intersectionLength = projection - halfChord;
	float3 intersection = ray * intersectionLength;
	float3 normal = normalize(intersection - spherePos);

	float3 pointLightPos = make_float3(sinf(time) * 3.0f, 2.0f, cosf(time) * 3.0f) + spherePos;

	float intensity = dot(normal, normalize(pointLightPos - intersection));

	if (intensity <= 0.0f)
		intensity = 0.0f;
	
	float4 result = { intensity, intensity, intensity, 1.0f };
	surf2Dwrite(result, surfObj, x * sizeof(result), y);
}

void CScene::UpdateTexture(float dt)
{
	m_time += dt;
	hipError_t error;

	dim3 blockDim(8, 8);
	dim3 gridDim((m_width - 1) / blockDim.x + 1, (m_height - 1) / blockDim.y + 1);

	hipGraphicsResource* cuResource;
	error = hipGraphicsGLRegisterImage(&cuResource, m_texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
	assert(!error);

	error = hipGraphicsMapResources(1, &cuResource);
	assert(!error);

	hipArray* cuArray;
	error = hipGraphicsSubResourceGetMappedArray(&cuArray, cuResource, 0, 0);
	assert(!error);

	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	hipSurfaceObject_t cuSurfaceObject;
	error = hipCreateSurfaceObject(&cuSurfaceObject, &resDesc);
	assert(!error);

	rayTracingKernel <<<gridDim, blockDim>>>(cuSurfaceObject, m_width, m_height, m_time);

	error = hipGetLastError();
	assert(!error);

	error = hipDeviceSynchronize();
	assert(!error);

	error = hipDestroySurfaceObject(cuSurfaceObject);
	assert(!error);

	error = hipGraphicsUnmapResources(1, &cuResource);
	assert(!error);

	error = hipGraphicsUnregisterResource(cuResource);
	assert(!error);
}