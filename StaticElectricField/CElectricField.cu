#include "hip/hip_runtime.h"
#include <GL\glew.h>

#include <hip/hip_runtime.h>
#include <>
#include <cuda_gl_interop.h>
#include <cassert>

#include "CElectricField.hpp"

static __global__ void computeElectricVectorFieldKernel(float3* grid, unsigned width, unsigned height, CElectricField::SParticle* particles, size_t count)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	float aspectRatio = float(width) / float(height);

	//get position coords from -1 to +1
	float2 pixelPosition =
	{
		((x / (float)(width - 1)) * 2.0f - 1.0f) * aspectRatio,
		(y / (float)(height - 1)) * 2.0f - 1.0f
	};

	float3 totalIntensity = {};

	for (size_t i = 0; i < count; ++i)
	{
		const CElectricField::SParticle p = particles[i];

		float dx = pixelPosition.x - p.position.x;
		float dy = pixelPosition.y - p.position.y;

		float distanceSqr = dx * dx + dy * dy;
		float invDistance = rsqrt(distanceSqr); // == 1 / distance

		if (distanceSqr < 0.001)
			continue;

		float scalarIntensity = p.charge / distanceSqr;

		float3 intensity =
		{
			scalarIntensity * (dx * invDistance),
			scalarIntensity * (dy * invDistance),
			//z component is for visual purpose only
			scalarIntensity
		};


		totalIntensity.x += intensity.x;
		totalIntensity.y += intensity.y;
		totalIntensity.z += intensity.z;
	}

	grid[x + y * width] = totalIntensity;
}

static __global__ void updateParticles(float3* grid, unsigned width, unsigned height, CElectricField::SParticle* particles, size_t count, float dt)
{
	auto index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= count) return;

	float aspectRatio = float(width) / float(height);

	CElectricField::SParticle& p = particles[index];
	int2 pixel =
	{
		(int)round((p.position.x / aspectRatio + 1.0f) * (width - 1) / 2.0f),
		(int)round((p.position.y + 1.0f) * (height - 1) / 2.0f)
	};

	if (pixel.x < 0)
	{
		p.velocity.x = abs(p.velocity.x);
		p.position.x = -aspectRatio;
		pixel.x = 0;
	}
	if (pixel.y < 0)
	{
		p.velocity.y = abs(p.velocity.y);
		p.position.y = -1.0f;
		pixel.y = 0;
	}

	if (pixel.x >= width)
	{
		pixel.x = width - 1;
		p.position.x = aspectRatio;
		p.velocity.x = -abs(p.velocity.x);
	}

	if (pixel.y >= height)
	{
		pixel.y = height - 1;
		p.position.y = 1.0f;
		p.velocity.y = -abs(p.velocity.y);
	}

	float3 intensity = grid[pixel.x + pixel.y * width];
	float2 force = { intensity.x * p.charge, intensity.y * p.charge };
	float2 accel = { force.x / p.mass, force.y / p.mass };

	if (abs(accel.x) > 100.0)
		accel.x = abs(accel.x) / accel.x * 100.0;

	if (abs(accel.y) > 100.0)
		accel.y = abs(accel.y) / accel.y * 100.0;

	p.velocity.x += accel.x * dt;
	p.velocity.y += accel.y * dt;

	p.position.x += p.velocity.x * dt;
	p.position.y += p.velocity.y * dt;
}

__global__ void renderFieldKernel(float3* grid, hipSurfaceObject_t surfObj, unsigned width, unsigned height)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		float3 intensity = grid[x + y * width];
		float field = intensity.z / 10.0f;

		float4 result = { 0, 0, 0, 1.0f };

		if (field >= 0.0f)
			result.x = field;
		else
			result.z = -field;

		surf2Dwrite(result, surfObj, x * sizeof(result), y);
	}
}

void CElectricField::UpdateState(float dt)
{
	hipError_t error;

	m_deviceVectorField.resize(m_width * m_height);

	dim3 blockDim(32, 32); //32*32 = 1024 threads per block
	dim3 gridDim((m_width - 1) / 32 + 1, (m_height - 1) / 32 + 1);

	computeElectricVectorFieldKernel <<<gridDim, blockDim >>> (m_deviceVectorField.data().get(), m_width, m_height, m_deviceParticles.data().get(), m_deviceParticles.size());
	error = hipGetLastError();
	assert(!error);

	updateParticles <<<1, unsigned(m_deviceParticles.size()) >>> (m_deviceVectorField.data().get(), m_width, m_height, m_deviceParticles.data().get(), m_deviceParticles.size(), dt);
	error = hipGetLastError();
	assert(!error);

	hipGraphicsResource* cuResource;
	error = hipGraphicsGLRegisterImage(&cuResource, m_texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
	assert(!error);

	error = hipGraphicsMapResources(1, &cuResource);
	assert(!error);

	hipArray* cuArray;
	error = hipGraphicsSubResourceGetMappedArray(&cuArray, cuResource, 0, 0);
	assert(!error);

	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	hipSurfaceObject_t cuSurfaceObject;
	error = hipCreateSurfaceObject(&cuSurfaceObject, &resDesc);
	assert(!error);

	renderFieldKernel <<<gridDim, blockDim >>> (m_deviceVectorField.data().get(), cuSurfaceObject, m_width, m_height);

	error = hipGetLastError();
	assert(!error);

	error = hipDeviceSynchronize();
	assert(!error);

	error = hipDestroySurfaceObject(cuSurfaceObject);
	assert(!error);

	error = hipGraphicsUnmapResources(1, &cuResource);
	assert(!error);

	error = hipGraphicsUnregisterResource(cuResource);
	assert(!error);
}