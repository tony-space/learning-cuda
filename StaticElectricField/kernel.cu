#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_texture_types.h>

#include "kernel.hpp"

__global__ void fetchKernel(float* output, hipTextureObject_t texObj, int width, int height)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;
	
	float u = x / (float)width;
	float v = y / (float)height;

	float4 pixel = tex2D<float4>(texObj, u, v);
	pixel.x *= 2;
	pixel.y *= 2;
	pixel.z *= 2;
	((float4*)output)[y * width + x] = pixel;
}

void TextureFetchTest()
{
	float pixels[] =
	{
		1.0f, 1.0f, 1.0f, 0.0f,
		1.0f, 0.0f, 0.0f, 0.0f,
		0.0f, 1.0f, 0.0f, 0.0f,
		0.0f, 0.0f, 1.0f, 0.0f
	};

	hipError_t error;

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	hipArray* cuArray;
	error = hipMallocArray(&cuArray, &channelDesc, 2, 2);
	error = hipMemcpyToArray(cuArray, 0, 0, pixels, sizeof(pixels), hipMemcpyHostToDevice);

	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	hipTextureDesc texDesc = {};
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	hipTextureObject_t texObj = 0;
	error = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

	float* d_output;
	error = hipMalloc(&d_output, 2 * 2 * 4 * sizeof(float));

	dim3 block(2, 2);
	fetchKernel <<<1, block >>> (d_output, texObj, 2, 2);
	error = hipGetLastError();
	error = hipDeviceSynchronize();
	
	for (float& p : pixels) p = 0.0f;

	error = hipMemcpy(pixels, d_output, sizeof(pixels), hipMemcpyDeviceToHost);

	error = hipFree(d_output);
	error = hipDestroyTextureObject(texObj);
	error = hipFreeArray(cuArray);
}