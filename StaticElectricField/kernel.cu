#include "hip/hip_runtime.h"
#include <GL\glew.h>

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_texture_types.h>
#include <cuda_gl_interop.h>
#include <hip/hip_math_constants.h>
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include <cassert>
#include "kernel.hpp"

__device__ static const float2 kElectronPosition = { -0.3f, 0.0f };
__device__ static const float2 kProtonPosition = { +0.3f, 0.0f };

thrust::device_vector<float> electricField;

__global__ void electricFieldKernel(float* grid, unsigned width, unsigned height)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		//get position coords from -1 to +1
		float2 pixelPosition =
		{
			(x / (float)(width - 1)) * 2.0f - 1.0f,
			(y / (float)(height - 1)) * 2.0f - 1.0f
		};

		float dx;
		float dy;
		float distanceSq;

		dx = kElectronPosition.x - pixelPosition.x;
		dy = kElectronPosition.y - pixelPosition.y;
		distanceSq = dx * dx + dy * dy;

		float electronField = -0.01 / distanceSq;

		if (distanceSq < 1e-4)
		{
			grid[x + y * width] = 0.0f;
			return;
		}


		dx = kProtonPosition.x - pixelPosition.x;
		dy = kProtonPosition.y - pixelPosition.y;
		distanceSq = dx * dx + dy * dy;

		if (distanceSq < 1e-4)
		{
			grid[x + y * width] = 0.0f;
			return;
		}

		float protonField = 1 / distanceSq;


		grid[x + y * width] = electronField + protonField;
	}
}

__global__ void renderFieldKernel(float* grid, hipSurfaceObject_t surfObj, unsigned width, unsigned height)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		float field = grid[x + y * width] / 3.0f;

		float4 result = { 0, 0, 0, 1.0f };

		if (field >= 0.0f)
			result.x = field;
		else
			result.z = -field;

		surf2Dwrite(result, surfObj, x * sizeof(result), y);
	}
}

void ProcessElectronField(unsigned textureId, unsigned width, unsigned height)
{
	if (electricField.size() != width * height)
		electricField.resize(width * height);

	hipError_t error;

	hipGraphicsResource* cuResource;
	error = hipGraphicsGLRegisterImage(&cuResource, textureId, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
	assert(!error);

	error = hipGraphicsMapResources(1, &cuResource);
	assert(!error);

	hipArray* cuArray;
	error = hipGraphicsSubResourceGetMappedArray(&cuArray, cuResource, 0, 0);
	assert(!error);

	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	hipSurfaceObject_t cuSurfaceObject;
	error = hipCreateSurfaceObject(&cuSurfaceObject, &resDesc);
	assert(!error);

	dim3 blockDim(32, 32); //32*32 = 1024 threads per block
	dim3 gridDim((width - 1) / 32 + 1, (height - 1) / 32 + 1);

	electricFieldKernel <<<gridDim, blockDim>>> (electricField.data().get(), width, height);
	error = hipGetLastError();
	assert(!error);

	/*auto pair = thrust::minmax_element(thrust::device, electricField.begin(), electricField.end());

	float* d_min = thrust::raw_pointer_cast(&(*pair.first));
	float* d_max = thrust::raw_pointer_cast(&(*pair.second));*/

	renderFieldKernel <<<gridDim, blockDim>>> (electricField.data().get(), cuSurfaceObject, width, height);
	error = hipGetLastError();
	assert(!error);

	error = hipDeviceSynchronize();
	assert(!error);

	error = hipDestroySurfaceObject(cuSurfaceObject);
	assert(!error);

	error = hipGraphicsUnmapResources(1, &cuResource);
	assert(!error);

	error = hipGraphicsUnregisterResource(cuResource);
	assert(!error);
}