#include "hip/hip_runtime.h"
#include <GL\glew.h>

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_texture_types.h>
#include <cuda_gl_interop.h>

#include "kernel.hpp"

__global__ void testFetchKernel(float* output, hipTextureObject_t texObj, int width, int height)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)width;
	float v = y / (float)height;

	float4 pixel = tex2D<float4>(texObj, u, v);
	pixel.x *= 2;
	pixel.y *= 2;
	pixel.z *= 2;
	((float4*)output)[y * width + x] = pixel;
}

void TextureFetchTest()
{
	float pixels[] =
	{
		1.0f, 1.0f, 1.0f, 0.0f,
		1.0f, 0.0f, 0.0f, 0.0f,
		0.0f, 1.0f, 0.0f, 0.0f,
		0.0f, 0.0f, 1.0f, 0.0f
	};

	hipError_t error;

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	hipArray* cuArray;
	error = hipMallocArray(&cuArray, &channelDesc, 2, 2);
	error = hipMemcpyToArray(cuArray, 0, 0, pixels, sizeof(pixels), hipMemcpyHostToDevice);

	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	hipTextureDesc texDesc = {};
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	hipTextureObject_t texObj = 0;
	error = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

	float* d_output;
	error = hipMalloc(&d_output, 2 * 2 * 4 * sizeof(float));

	dim3 block(2, 2);
	testFetchKernel <<<1, block >>> (d_output, texObj, 2, 2);
	error = hipGetLastError();
	error = hipDeviceSynchronize();

	for (float& p : pixels) p = 0.0f;

	error = hipMemcpy(pixels, d_output, sizeof(pixels), hipMemcpyDeviceToHost);

	error = hipFree(d_output);
	error = hipDestroyTextureObject(texObj);
	error = hipFreeArray(cuArray);
}


__global__ void testOpenGLFetchKernel(unsigned char* output, hipTextureObject_t texObj, int width, int height)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)width;
	float v = y / (float)height;

	char4 pixel = tex2D<char4>(texObj, u, v);
	((char4*)output)[y * width + x] = pixel;
}

void OpenGLTextureFetchTest(unsigned textureId)
{
	hipError_t error;

	hipGraphicsResource* cuResource;
	error = hipGraphicsGLRegisterImage(&cuResource, textureId, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);

	error = hipGraphicsMapResources(1, &cuResource);

	hipArray* cuArray;
	error = hipGraphicsSubResourceGetMappedArray(&cuArray, cuResource, 0, 0);

	hipChannelFormatDesc channelDesc;
	error = hipGetChannelDesc(&channelDesc, cuArray);

	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	hipTextureDesc texDesc = {};
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	hipTextureObject_t texObj = 0;
	error = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

	unsigned char* d_output;
	error = hipMalloc(&d_output, 2 * 2 * 4 * sizeof(float));

	unsigned char pixels[2 * 2 * 4] = {};

	dim3 block(2, 2);
	testOpenGLFetchKernel <<<1, block >>>(d_output, texObj, 2, 2);
	error = hipGetLastError();
	error = hipDeviceSynchronize();

	error = hipMemcpy(pixels, d_output, sizeof(pixels), hipMemcpyDeviceToHost);

	error = hipFree(d_output);
	error = hipDestroyTextureObject(texObj);
	error = hipGraphicsUnmapResources(1, &cuResource);
	error = hipGraphicsUnregisterResource(cuResource);
}