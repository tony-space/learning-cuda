#include "hip/hip_runtime.h"
#include <GL\glew.h>

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_texture_types.h>
#include <cuda_gl_interop.h>
#include <hip/hip_math_constants.h>

#include <cassert>
#include <chrono>

#include "kernel.hpp"

__global__ void textureFetchKernel(float* output, hipTextureObject_t texObj, int width, int height)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)width;
	float v = y / (float)height;

	float4 pixel = tex2D<float4>(texObj, u, v);
	pixel.x *= 2;
	pixel.y *= 2;
	pixel.z *= 2;
	((float4*)output)[y * width + x] = pixel;
}

void TextureFetchTest()
{
	float pixels[] =
	{
		1.0f, 1.0f, 1.0f, 0.0f,
		1.0f, 0.0f, 0.0f, 0.0f,
		0.0f, 1.0f, 0.0f, 0.0f,
		0.0f, 0.0f, 1.0f, 0.0f
	};

	hipError_t error;

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	hipArray* cuArray;
	error = hipMallocArray(&cuArray, &channelDesc, 2, 2);
	assert(!error);

	error = hipMemcpyToArray(cuArray, 0, 0, pixels, sizeof(pixels), hipMemcpyHostToDevice);
	assert(!error);

	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	hipTextureDesc texDesc = {};
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	hipTextureObject_t texObj = 0;
	error = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);
	assert(!error);

	float* d_output;
	error = hipMalloc(&d_output, 2 * 2 * 4 * sizeof(float));
	assert(!error);

	dim3 block(2, 2);
	textureFetchKernel <<<1, block>>> (d_output, texObj, 2, 2);
	error = hipGetLastError();
	assert(!error);

	error = hipDeviceSynchronize();
	assert(!error);

	for (float& p : pixels) p = 0.0f;

	error = hipMemcpy(pixels, d_output, sizeof(pixels), hipMemcpyDeviceToHost);
	assert(!error);

	error = hipFree(d_output);
	assert(!error);

	error = hipDestroyTextureObject(texObj);
	assert(!error);

	error = hipFreeArray(cuArray);
	assert(!error);
}


__global__ void openGLTextureFetchKernel(unsigned char* output, hipTextureObject_t texObj, unsigned width, unsigned height)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)width;
	float v = y / (float)height;

	char4 pixel = tex2D<char4>(texObj, u, v);
	((char4*)output)[y * width + x] = pixel;
}

void OpenGLTextureFetchTest(unsigned textureId)
{
	hipError_t error;

	hipGraphicsResource* cuResource;
	error = hipGraphicsGLRegisterImage(&cuResource, textureId, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
	assert(!error);

	error = hipGraphicsMapResources(1, &cuResource);
	assert(!error);

	hipArray* cuArray;
	error = hipGraphicsSubResourceGetMappedArray(&cuArray, cuResource, 0, 0);
	assert(!error);

	hipChannelFormatDesc channelDesc;
	error = hipGetChannelDesc(&channelDesc, cuArray);
	assert(!error);

	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	hipTextureDesc texDesc = {};
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	hipTextureObject_t texObj = 0;
	error = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);
	assert(!error);

	unsigned char* d_output;
	error = hipMalloc(&d_output, 2 * 2 * 4 * sizeof(float));
	assert(!error);

	unsigned char pixels[2 * 2 * 4] = {};

	dim3 blockDim(2, 2);
	openGLTextureFetchKernel <<<1, blockDim>>> (d_output, texObj, 2, 2);
	error = hipGetLastError();
	assert(!error);

	error = hipDeviceSynchronize();
	assert(!error);

	error = hipMemcpy(pixels, d_output, sizeof(pixels), hipMemcpyDeviceToHost);
	assert(!error);

	error = hipFree(d_output);
	assert(!error);

	error = hipDestroyTextureObject(texObj);
	assert(!error);

	error = hipGraphicsUnmapResources(1, &cuResource);
	assert(!error);

	error = hipGraphicsUnregisterResource(cuResource);
	assert(!error);
}

__global__ void pboGeneratorKernel(float4* storage, unsigned width, unsigned height, float time)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)width;
	float v = y / (float)height;

	float4 result = {
		cos(u * HIP_PI_F * 2.0f * time + time) / 2.0f + 0.5f,
		0,
		sin(v * HIP_PI_F * 2.0f * time + time) / 2.0f + 0.5f,
		1 };

	storage[y * width + x] = result;
}

void GeneratePBO(unsigned pboUnpackedBuffer, unsigned width, unsigned height)
{
	hipError_t error;
	hipGraphicsResource* cuResource;

	error = hipGraphicsGLRegisterBuffer(&cuResource, pboUnpackedBuffer, hipGraphicsRegisterFlagsWriteDiscard);
	assert(!error);

	error = hipGraphicsMapResources(1, &cuResource);
	assert(!error);

	void* d_pboStorage;
	size_t storageSize;
	error = hipGraphicsResourceGetMappedPointer(&d_pboStorage, &storageSize, cuResource);
	assert(!error);

	static auto startTime = std::chrono::system_clock::now();
	auto now = std::chrono::system_clock::now();
	auto delta = now - startTime;
	auto milliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(delta);
	float time = milliseconds.count() / 1000.0f;

	dim3 gridDim(width / 32, height / 32);
	dim3 blockDim(32, 32);
	pboGeneratorKernel <<<gridDim, blockDim>>> ((float4*)d_pboStorage, width, height, time);

	error = hipGetLastError();
	assert(!error);

	error = hipDeviceSynchronize();
	assert(!error);

	error = hipGraphicsUnmapResources(1, &cuResource);
	assert(!error);

	error = hipGraphicsUnregisterResource(cuResource);
	assert(!error);
}


__global__ void openGlTextureModifier(hipSurfaceObject_t surfObj, unsigned width, unsigned height, float time)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)width;
	float v = y / (float)height;

	float value1 = sin(v * HIP_PI_F * 2.0f * time + time) / 2.0f + 0.5f;
	float value2 = cos(u * HIP_PI_F * 2.0f * time + time) / 2.0f + 0.5f;
	float value3 = value1 + value2;

	float4 result = {
		value1,
		value2,
		value3,
		1 };

	surf2Dwrite(result, surfObj, x * sizeof(result), y);
}

void ModifyTexture(unsigned textureId, unsigned width, unsigned height)
{
	hipError_t error;

	hipGraphicsResource* cuResource;
	error = hipGraphicsGLRegisterImage(&cuResource, textureId, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
	assert(!error);

	error = hipGraphicsMapResources(1, &cuResource);
	assert(!error);

	hipArray* cuArray;
	error = hipGraphicsSubResourceGetMappedArray(&cuArray, cuResource, 0, 0);
	assert(!error);

	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	hipSurfaceObject_t cuSurfaceObject;
	error = hipCreateSurfaceObject(&cuSurfaceObject, &resDesc);
	assert(!error);
	
	static auto startTime = std::chrono::system_clock::now();
	auto now = std::chrono::system_clock::now();
	auto delta = now - startTime;
	auto milliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(delta);
	float time = milliseconds.count() / 1000.0f;

	dim3 gridDim(width / 32, height / 32);
	dim3 blockDim(32, 32);
	openGlTextureModifier <<<gridDim, blockDim>>> (cuSurfaceObject, width, height, time);

	error = hipGetLastError();
	assert(!error);

	error = hipDeviceSynchronize();
	assert(!error);

	error = hipDestroySurfaceObject(cuSurfaceObject);
	assert(!error);

	error = hipGraphicsUnmapResources(1, &cuResource);
	assert(!error);

	error = hipGraphicsUnregisterResource(cuResource);
	assert(!error);
}